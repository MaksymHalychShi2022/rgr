#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include <fstream>
#include <iostream>
#include <string>
#include <csignal>
#include <fstream>
#include <sstream>
#include <cmath>

#include "image.h"
#include "results.h"

__global__ void dwt2d_kernel(
        float *signal,
        float *tmp_row,
        float *tmp_col,
        int rows,
        int cols,
        int half_rows,
        int half_cols,
        float norm
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < rows && idy < half_cols) {
        // Apply 1D Haar transform to each row
        tmp_row[idx * cols + idy] = (signal[idx * cols + 2 * idy] + signal[idx * cols + 2 * idy + 1]) / norm;
        tmp_row[idx * cols + half_cols + idy] =
                (signal[idx * cols + 2 * idy] - signal[idx * cols + 2 * idy + 1]) / norm;
    }

    __syncthreads();

    if (idx < half_rows && idy < cols) {
        // Apply 1D Haar transform to each column
        tmp_col[idy * rows + idx] = (tmp_row[2 * idx * cols + idy] + tmp_row[(2 * idx + 1) * cols + idy]) / norm;
        tmp_col[idy * rows + half_rows + idx] =
                (tmp_row[2 * idx * cols + idy] - tmp_row[(2 * idx + 1) * cols + idy]) / norm;
    }
}

void dwt2d(
        float *host_signal,
        int rows,
        int cols,
) {
    float *dev_signal, *dev_tmp_row, *dev_tmp_col;
    float norm = sqrt(2);
    int half_rows = rows / 2;
    int half_cols = cols / 2;
    size_t size = rows * cols * sizeof(float);

    hipMalloc(&dev_signal, size);
    hipMalloc(&dev_tmp_row, size);
    hipMalloc(&dev_tmp_col, size);

    hipMemcpy(dev_signal, host_signal, size, hipMemcpyHostToDevice);

    dim3 blocks(16, 16);
    dim3 grid((cols + blocks.x - 1) / blocks.x, (rows + blocks.y - 1) / blocks.y);

    dwt2d_kernel<<<grid, blocks>>>(dev_signal, dev_tmp_row, dev_tmp_col, rows, cols, half_rows, half_cols, norm);
    hipMemcpy(dev_signal, dev_tmp_col, size, hipMemcpyDeviceToDevice);
    hipMemcpy(host_signal, dev_signal, size, hipMemcpyDeviceToHost);

    hipFree(dev_signal);
    hipFree(dev_tmp_row);
    hipFree(dev_tmp_col);
}


void test_gpu_time(std::vector <std::string> test_set) {
    auto results_dwt = Results();
    std::chrono::high_resolution_clock::time_point start;
    std::chrono::high_resolution_clock::time_point stop;
    size_t duration;

    for (auto test: test_set) {
        int rows, cols;
        float *host_signal;
        load_image(host_signal, rows, cols, test);

        start = std::chrono::high_resolution_clock::now();
        dwt2d(host_signal, rows, cols);
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
        results_dwt.addRecord(rows, cols, "gpu", duration);
        delete[] host_signal;
    }

    results_dwt.writeToCSV("../dwt_results.csv");
}

void test_gpu_certainty(const std::string &output) {
    int rows = 8, cols = 8;
    float *host_signal = new float[rows * cols];
    for (auto i = 0; i < rows * cols; i++) {
        host_signal[i] = i;
    }

    dwt2d(host_signal, rows, cols);
    write_image(host_signal, rows, cols, output);
    delete[] host_signal;
}

int main() {
    std::vector <std::string> test_set = {
            "../data/test_big.tsv",
            "../data/test_medium.tsv",
            "../data/test_small.tsv",
    };

    test_gpu_time(test_set);
    test_gpu_certainty("../data/test_certainty_gpu_output.tsv");

    return 0;
}
